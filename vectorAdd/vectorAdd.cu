
#include <hip/hip_runtime.h>
__global__ void initArray(float* arr, int N)
{
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < N)
    {
        arr[id] = id;
    }
}

__global__ void addVectors(float* first, float* second, float* result, int N)
{
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < N)
    {
        result[id] = first[id] + second[id];
    }
}

int main()
{
    int N = 1 << 20;

    float *a, *b, *c;

    hipMallocManaged(&a, N*sizeof(float));
    hipMallocManaged(&b, N*sizeof(float));
    hipMallocManaged(&c, N*sizeof(float));

    int numThreads = 1<<9;
    int numBlocks = N/numThreads;
    initArray<<<numBlocks, numThreads>>> (a, N);
    initArray<<<numBlocks, numThreads>>> (b, N);

    hipDeviceSynchronize();

    addVectors<<<numBlocks, numThreads>>> (a, b, c, N);

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(c);
}
