
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

#define ROW_SIZE 1024
#define COLUMN_SIZE 1024
#define BLK_SIZE 512
#define MIN 1
#define MAX 100

__global__ void setup_kernel(hiprandState *state, unsigned long long seed){

  int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if(idx < COLUMN_SIZE)
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void init_matrix(int *matrix, int dim, int min, int max, hiprandState *state)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;

    int row = idx / dim;
    float myrandf = hiprand_uniform(&state[row]);
    myrandf *= (max - min+0.999999);
    myrandf += min;
    int myrand = (int)truncf(myrandf);

    matrix[idx] = myrand;
}

int main()
{
    hiprandState *d_state;
    hipMalloc(&d_state, COLUMN_SIZE * sizeof(hiprandState));
    setup_kernel<<<2, BLK_SIZE>>> (d_state, clock());

    int *d_matrix;
    hipMalloc(&d_matrix, (COLUMN_SIZE * ROW_SIZE) * sizeof(int));

    init_matrix<<<(BLK_SIZE * 4), BLK_SIZE>>> (d_matrix, ROW_SIZE, MIN, MAX, d_state);
}